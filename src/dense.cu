#include "hip/hip_runtime.h"
#include "dense.hpp"
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void replicate_bias_kernel(const float *bias, float *out, int batch_size, int output_dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * output_dim;

    if (idx < total) {
        int j = idx % output_dim;
        out[idx] = bias[j];
    }
}

Dense::Dense(int input_dim, int output_dim, Device device) : input_dim(input_dim), output_dim(output_dim) {
    // Create on CPU regardless of the intended device.
    Tensor w(input_dim * output_dim, CPU);

    // Initialize on CPU.
    float limit = sqrt(6.0f / (input_dim + output_dim));
    for (size_t i = 0; i < w.size(); ++i) {
        w.data()[i] = ((float)rand() / RAND_MAX * 2.0f - 1.0f) * limit;
    }

    // Transfer to CUDA if needed.
    if (device == CUDA) {
        w.toCUDA();
    }

    weight = make_shared<Variable>(w, true, "Dense_weight");

    Tensor b(output_dim, device);
    b.fill(0.0f);
    bias = make_shared<Variable>(b, true, "Dense_bias");
}

VarPtr Dense::forward(const VarPtr &input) {
    int batch_size = input->data.size() / input_dim;
    auto z = MatMulFunction::apply(input, weight, batch_size, input_dim, output_dim);

    auto out = AddFunction::apply(z, bias);

    return out;
}

vector<VarPtr> Dense::parameters() {
    return {weight, bias};
}