#include "hip/hip_runtime.h"
#include "softmax.hpp"

Softmax::Softmax(int batch_size, int num_classes)
    : batch_size(batch_size), num_classes(num_classes) {}

VarPtr Softmax::forward(const VarPtr &input) {
    return SoftmaxFunction::apply(input, batch_size, num_classes);
}

__global__ void softmax_forward_kernel(const float *input, float *output, int num_classes) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int index = blockIdx.x * num_classes + tid; // one sample per block
    float val = input[index];
    sdata[tid] = val;
    __syncthreads();

    // Compute next power of two greater than or equal to num_classes.
    int n = num_classes;
    int s = 1;
    while (s < n)
        s *= 2;

    // Reduction to compute maximum over valid elements.
    for (int stride = s / 2; stride > 0; stride /= 2) {
        if (tid < stride && (tid + stride) < n) {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + stride]);
        }
        __syncthreads();
    }
    float max_val = sdata[0];

    // Compute exp(x - max_val) for numerical stability.
    float exp_val = expf(val - max_val);
    sdata[tid] = exp_val;
    __syncthreads();

    // Reduction to compute the sum of exponentials.
    for (int stride = s / 2; stride > 0; stride /= 2) {
        if (tid < stride && (tid + stride) < n) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }
    float sum_val = sdata[0];
    output[index] = exp_val / sum_val;
}

Tensor softmax_forward(const Tensor &input, int batch_size, int num_classes) {
    Tensor output(input.size(), input.device());

    if (input.device() == CPU) {
        // CPU branch.
        const float *in_data = input.data();
        float *out_data = output.data();
        for (int b = 0; b < batch_size; ++b) {
            int offset = b * num_classes;
            // Find max for numerical stability
            float max_val = in_data[offset];
            for (int i = 1; i < num_classes; ++i) {
                max_val = max(max_val, in_data[offset + i]);
            }
            // Compute exponentials and accumulate their sum
            float sum_exp = 0.0f;
            for (int i = 0; i < num_classes; ++i) {
                float exp_val = exp(in_data[offset + i] - max_val);
                out_data[offset + i] = exp_val;
                sum_exp += exp_val;
            }
            // Normalize.
            for (int i = 0; i < num_classes; ++i) {
                out_data[offset + i] /= sum_exp;
            }
        }
    } else {
        // CUDA branch.
        const float *in_ptr = input.data(); // device pointer
        float *out_ptr = output.data();     // device pointer

        // Launch one block per sample with num_classes threads per block.
        dim3 gridSize(batch_size);
        dim3 blockSize(num_classes);
        size_t sharedMemSize = num_classes * sizeof(float);
        // Kernel launch (defined below).
        softmax_forward_kernel<<<gridSize, blockSize, sharedMemSize>>>(in_ptr, out_ptr, num_classes);
        hipDeviceSynchronize();
    }

    return output;
}

VarPtr SoftmaxFunction::apply(const VarPtr &input, int batch_size, int num_classes) {
    auto func = make_shared<SoftmaxFunction>();
    func->batch_size = batch_size;
    func->num_classes = num_classes;
    func->saved_input = input;

    // Call the unified softmax forward helper.
    Tensor out_tensor = softmax_forward(input->data, batch_size, num_classes);
    func->softmax_output = out_tensor; // Save computed output for backward.

    auto out = make_shared<Variable>(out_tensor, input->requires_grad, "Softmax_out");
    out->set_creator(func);
    func->inputs.push_back(input);
    func->output = out;

    return out;
}

// Softmax backward: compute dL/dx using:
//   dL/dx_i = y_i * (dL/dy_i - sum_j (dL/dy_j * y_j))
vector<Tensor> SoftmaxFunction::backward(const Tensor &grad_output) {
    // Ensure backward is computed on CPU.
    Tensor grad_out_cpu = grad_output;
    if (grad_output.device() != CPU) {
        grad_out_cpu.toCPU();
    }
    Tensor y = softmax_output;
    if (y.device() != CPU) {
        y.toCPU();
    }
    int total = batch_size * num_classes;
    Tensor grad_input(total, CPU);
    float *grad_in_ptr = grad_input.data();
    const float *grad_out_ptr = grad_out_cpu.data();
    const float *y_ptr = y.data();

    for (int b = 0; b < batch_size; ++b) {
        int offset = b * num_classes;
        float dot = 0.0f;
        for (int j = 0; j < num_classes; ++j) {
            dot += grad_out_ptr[offset + j] * y_ptr[offset + j];
        }
        for (int i = 0; i < num_classes; ++i) {
            grad_in_ptr[offset + i] = y_ptr[offset + i] * (grad_out_ptr[offset + i] - dot);
        }
    }

    // Convert grad_input back to CUDA if needed.
    if (grad_output.device() != CPU) {
        grad_input.toCUDA();
    }

    return {grad_input};
}
