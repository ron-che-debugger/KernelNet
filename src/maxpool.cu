#include "hip/hip_runtime.h"
#include "maxpool.hpp"
namespace kernelnet {
namespace nn {
/**
 * @brief Constructor for the MaxPool2D module.
 *
 * Initializes pooling parameters and input dimensions.
 *
 * @param kernel_size Size of the square pooling window.
 * @param stride Stride used for pooling.
 * @param batch_size Number of samples in the batch.
 * @param channels Number of channels in the input.
 * @param input_height Height of the input image.
 * @param input_width Width of the input image.
 */
MaxPool2D::MaxPool2D(int kernel_size, int stride,
                     int batch_size, int channels,
                     int input_height, int input_width)
    : kernel_size(kernel_size), stride(stride),
      batch_size(batch_size), channels(channels),
      input_height(input_height), input_width(input_width) {}

/**
 * @brief Forward pass for MaxPool2D.
 *
 * Applies max pooling on the input variable and returns the pooled output.
 *
 * @param input Input variable.
 * @return Output variable after max pooling.
 */
VarPtr MaxPool2D::forward(const VarPtr &input) {
    return MaxPool2DFunction::apply(input, batch_size, channels, input_height, input_width, kernel_size, stride);
}

/**
 * @brief CUDA kernel for the forward pass of max pooling.
 *
 * For each output element, finds the maximum value over the corresponding
 * pooling window and stores the index of the maximum element.
 *
 * @param in_data Pointer to the input tensor data.
 * @param out_data Pointer to the output tensor data.
 * @param d_max_indices Pointer to an array to store indices of max values.
 * @param batch_size Number of samples in the batch.
 * @param channels Number of channels in the input.
 * @param input_height Height of the input image.
 * @param input_width Width of the input image.
 * @param kernel_size Size of the square pooling window.
 * @param stride Stride used for pooling.
 * @param output_height Calculated height of the output.
 * @param output_width Calculated width of the output.
 */
__global__ void maxpool_forward_kernel(const float *in_data, float *out_data, int *d_max_indices,
                                       int batch_size, int channels, int input_height, int input_width,
                                       int kernel_size, int stride, int output_height, int output_width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = batch_size * channels * output_height * output_width;
    if (idx < total) {
        // Decode output index into b, c, oh, ow.
        int ow = idx % output_width;
        int tmp = idx / output_width;
        int oh = tmp % output_height;
        tmp = tmp / output_height;
        int c = tmp % channels;
        int b = tmp / channels;

        float max_val = -FLT_MAX;
        int max_idx = -1;
        int start_h = oh * stride;
        int start_w = ow * stride;

        for (int i = 0; i < kernel_size; ++i) {
            for (int j = 0; j < kernel_size; ++j) {
                int ih = start_h + i;
                int iw = start_w + j;
                int in_index = b * (channels * input_height * input_width) +
                               c * (input_height * input_width) +
                               ih * input_width + iw;
                float val = in_data[in_index];
                if (val > max_val) {
                    max_val = val;
                    max_idx = in_index;
                }
            }
        }
        out_data[idx] = max_val;
        d_max_indices[idx] = max_idx;
    }
}

/**
 * @brief Helper function for the forward pass of max pooling.
 *
 * Computes output dimensions, allocates the output tensor, and performs max pooling
 * on CPU or CUDA. Also returns the indices of maximum values for use in the backward pass.
 *
 * @param input Input tensor.
 * @param batch_size Number of samples in the batch.
 * @param channels Number of channels in the input.
 * @param input_height Height of the input image.
 * @param input_width Width of the input image.
 * @param kernel_size Size of the pooling kernel.
 * @param stride Pooling stride.
 * @param output_height (Output) Calculated output height.
 * @param output_width (Output) Calculated output width.
 * @param max_indices (Output) Vector to store indices of maximum elements.
 * @return The output tensor after max pooling.
 */
Tensor maxpool_forward(const Tensor &input, int batch_size, int channels,
                       int input_height, int input_width,
                       int kernel_size, int stride,
                       int &output_height, int &output_width,
                       vector<int> &max_indices) {
    // Compute output dimensions.
    output_height = (input_height - kernel_size) / stride + 1;
    output_width = (input_width - kernel_size) / stride + 1;
    size_t output_size = batch_size * channels * output_height * output_width;

    // Allocate the output tensor on the same device as the input.
    Tensor out_tensor(output_size, input.device());
    max_indices.resize(output_size, -1);

    if (input.device() == CPU) {
        // CPU branch.
        const float *in_data = input.data();
        float *out_data = out_tensor.data();

        for (int b = 0; b < batch_size; ++b) {
            for (int c = 0; c < channels; ++c) {
                for (int oh = 0; oh < output_height; ++oh) {
                    for (int ow = 0; ow < output_width; ++ow) {
                        int out_index = b * (channels * output_height * output_width) +
                                        c * (output_height * output_width) +
                                        oh * output_width + ow;
                        float max_val = -FLT_MAX;
                        int max_idx = -1;
                        int start_h = oh * stride;
                        int start_w = ow * stride;
                        for (int i = 0; i < kernel_size; ++i) {
                            for (int j = 0; j < kernel_size; ++j) {
                                int ih = start_h + i;
                                int iw = start_w + j;
                                int in_index = b * (channels * input_height * input_width) +
                                               c * (input_height * input_width) +
                                               ih * input_width + iw;
                                float val = in_data[in_index];
                                if (val > max_val) {
                                    max_val = val;
                                    max_idx = in_index;
                                }
                            }
                        }
                        out_data[out_index] = max_val;
                        max_indices[out_index] = max_idx;
                    }
                }
            }
        }
    } else {
        // CUDA branch.
        int *d_max_indices;
        hipMalloc(&d_max_indices, output_size * sizeof(int));

        const float *in_data = input.data(); // device pointer
        float *out_data = out_tensor.data(); // device pointer

        dim3 blockSize(256);
        dim3 gridSize((output_size + blockSize.x - 1) / blockSize.x);
        maxpool_forward_kernel<<<gridSize, blockSize>>>(in_data, out_data, d_max_indices,
                                                        batch_size, channels, input_height, input_width,
                                                        kernel_size, stride, output_height, output_width);
        hipDeviceSynchronize();

        // Copy the computed indices from device to host.
        hipMemcpy(max_indices.data(), d_max_indices, output_size * sizeof(int), hipMemcpyDeviceToHost);
        hipFree(d_max_indices);
    }

    return out_tensor;
}

/**
 * @brief Applies the MaxPool2DFunction forward pass.
 *
 * Sets up pooling parameters and calls the helper function to perform max pooling.
 *
 * @param input Input variable.
 * @param batch_size Number of samples in the batch.
 * @param channels Number of channels in the input.
 * @param input_height Height of the input image.
 * @param input_width Width of the input image.
 * @param kernel_size Size of the pooling kernel.
 * @param stride Pooling stride.
 * @return Output variable after max pooling.
 */
VarPtr MaxPool2DFunction::apply(const VarPtr &input,
                                int batch_size, int channels,
                                int input_height, int input_width,
                                int kernel_size, int stride) {
    auto func = make_shared<MaxPool2DFunction>();
    func->saved_input = input;
    func->batch_size = batch_size;
    func->channels = channels;
    func->input_height = input_height;
    func->input_width = input_width;
    func->kernel_size = kernel_size;
    func->stride = stride;

    int output_height, output_width;
    vector<int> max_indices;
    // Call the helper forward function.
    Tensor out_tensor = maxpool_forward(input->data, batch_size, channels,
                                        input_height, input_width,
                                        kernel_size, stride,
                                        output_height, output_width,
                                        max_indices);
    func->output_height = output_height;
    func->output_width = output_width;
    func->max_indices = max_indices;

    auto out = make_shared<Variable>(out_tensor, input->requires_grad, "Maxpool_out");
    out->set_creator(func);
    func->inputs.push_back(input);
    func->output = out;

    return out;
}

/**
 * @brief CUDA kernel for the backward pass of max pooling.
 *
 * For each output element, uses the stored index of the maximum input element
 * to propagate the gradient.
 *
 * @param grad_out Pointer to the gradient output tensor.
 * @param grad_in Pointer to the gradient input tensor (to be updated).
 * @param max_indices Pointer to the array of indices stored during forward.
 * @param output_size Total number of output elements.
 */
__global__ void maxpool_backward_kernel(const float *grad_out, float *grad_in, const int *max_indices, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        int in_idx = max_indices[idx];
        atomicAdd(&grad_in[in_idx], grad_out[idx]);
    }
}

/**
 * @brief Computes the backward pass of the MaxPool2DFunction.
 *
 * Propagates gradients from the output back to the input using the stored max indices.
 *
 * @param grad_output Gradient tensor from the next layer.
 * @return A vector containing a single tensor representing the gradient with respect to the input.
 */
vector<Tensor> MaxPool2DFunction::backward(const Tensor &grad_output) {
    if (grad_output.device() == CPU) {
        Tensor grad_input(batch_size * channels * input_height * input_width, CPU);
        grad_input.fill(0.0f);
        int output_size = batch_size * channels * output_height * output_width;
        const float *grad_out_data = grad_output.data();
        float *grad_in_data = grad_input.data();
        for (int idx = 0; idx < output_size; ++idx) {
            int in_idx = max_indices[idx];
            grad_in_data[in_idx] += grad_out_data[idx];
        }
        return {grad_input};
    } else {
        // CUDA branch.
        int output_size = batch_size * channels * output_height * output_width;
        // Allocate grad_input with size = input size.
        Tensor grad_input(batch_size * channels * input_height * input_width, CUDA);
        grad_input.fill(0.0f);

        // Allocate device memory for max_indices.
        int *d_max_indices;
        hipMalloc(&d_max_indices, output_size * sizeof(int));
        hipMemcpy(d_max_indices, max_indices.data(), output_size * sizeof(int), hipMemcpyHostToDevice);

        // Launch the kernel to compute gradients.
        dim3 blockSize(256);
        dim3 gridSize((output_size + blockSize.x - 1) / blockSize.x);
        maxpool_backward_kernel<<<gridSize, blockSize>>>(grad_output.data(), grad_input.data(), d_max_indices, output_size);
        hipDeviceSynchronize();
        hipFree(d_max_indices);

        return {grad_input};
    }
}
} // namespace nn
} // namespace kernelnet